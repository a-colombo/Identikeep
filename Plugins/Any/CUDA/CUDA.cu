#include "CUDA.h"

// Logging
#include "plog/Log.h"


bool CUDA::Collect(int argc, char *argv[])
{
  bool is_ok = true;

  LOG_VERBOSE << "Parsing " << this->Name() << " info...";
  
  int nDevices;

  hipGetDeviceCount(&nDevices);

  hipDeviceProp_t prop[nDevices];

  std::vector<std::string> 	dev_list;
  std::vector<float> 	   	totalGlobalMem_list;
  std::vector<int> 		clockRate_list;
  std::vector<std::string> 	version_list;

  for (int i = 0; i < nDevices; i++){
	hipGetDeviceProperties(&prop[i],i);

        dev_list.push_back(prop[i].name);
	totalGlobalMem_list.push_back(double(prop[i].totalGlobalMem)/1000000.);
	clockRate_list.push_back(prop[i].clockRate);
	
	std::string vv=std::to_string(prop[i].major)+"."+std::to_string(prop[i].minor);
	version_list.push_back(vv);
  }


     
    Item<std::string> idev_list = Item<std::string>("cudaDevice", match::list | match::node | match::world, dev_list);
    m_items.strings.push_back(idev_list);

    Item<std::string> iversion = Item<std::string>("cudaVersion", match::list | match::node | match::world, version_list);
    m_items.strings.push_back(iversion);

    Item<int> iclock = Item<int>("cudaClock", "KHz", match::list | match::node | match::world, clockRate_list);
    m_items.integers.push_back(iclock);

    Item<float> imem = Item<float>("cudaTotMem", "MB", match::list | match::node | match::world, totalGlobalMem_list);
    m_items.floats.push_back(imem);



    return is_ok;
};
