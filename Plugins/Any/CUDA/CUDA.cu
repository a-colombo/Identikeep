#include "CUDA.h"

// Logging
#include "plog/Log.h"


bool CUDA::Collect(int argc, char *argv[])
{
  bool is_ok = true;

  LOG_VERBOSE << "Parsing " << this->Name() << " info...";
  
  int nDevices=0;

  hipGetDeviceCount(&nDevices);
  
  if(nDevices>0){

    hipDeviceProp_t prop[nDevices];
    int runtimeVersion=0;
    int driverVersion=0;
    
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    
    std::vector<std::string> 	dev_list;
    std::vector<float> 	   	totalGlobalMem_list;
    std::vector<int> 		clockRate_list;
    std::vector<std::string> 	compcap_list;

    for (int i = 0; i < nDevices; i++){
        hipGetDeviceProperties(&prop[i],i);

            dev_list.push_back(prop[i].name);
        totalGlobalMem_list.push_back(double(prop[i].totalGlobalMem)/1000000.);
        clockRate_list.push_back(prop[i].clockRate);
        
        std::string vv=std::to_string(prop[i].major)+"."+std::to_string(prop[i].minor);
        compcap_list.push_back(vv);
    }


        
        Item<std::string> idev_list = Item<std::string>("cudaDevice", match::list | match::node | match::world, dev_list);
        m_items.strings.push_back(idev_list);

        Item<std::string> icompcap = Item<std::string>("cudaComputeCapability", match::list | match::node | match::world, compcap_list);
        m_items.strings.push_back(icompcap);

        Item<int> iclock = Item<int>("cudaClock", "KHz", match::list | match::node | match::world, clockRate_list);
        m_items.integers.push_back(iclock);

        Item<float> imem = Item<float>("cudaTotMem", "MB", match::list| match::node | match::world, totalGlobalMem_list);
        m_items.floats.push_back(imem);
        
        Item<int> iDriverVersion = Item<int>("cudaDriverVersion",  match::list | match::node | match::world, driverVersion);
        m_items.integers.push_back(iDriverVersion);
        
        Item<int> iRuntimeVersion = Item<int>("cudaRuntimeVersion",  match::list | match::node | match::world, runtimeVersion);
        m_items.integers.push_back(iRuntimeVersion);
        
  }
  else{
        Item<std::string> idev_list = Item<std::string>("cudaDevice", match::list | match::node | match::world, "No NVIDIA GPU detected");
        m_items.strings.push_back(idev_list);      
      
}


    return is_ok;
};
